#include "buffer.cuh"
#include "nvshmem.hpp"
#include "sync.cuh"
#include "utils.hpp"

namespace nvshmem_tutorial {

void Buffer::internode_all_gather(std::vector<torch::Tensor>& tensor_list,
                                  const torch::Tensor& tensor, bool async_op) {
  if (!tensor.is_cuda()) {
    throw std::runtime_error("internode_all_gather expects CUDA tensor");
  }

  if (buffer_ptrs_[nvl_rank_] == nullptr) {
    throw std::runtime_error("Local NVLink buffer not allocated");
  }

  CUDA_CHECK(hipMemcpyAsync(buffer_ptrs_[nvl_rank_], tensor.data_ptr(),
                             tensor.nbytes(), hipMemcpyDeviceToDevice,
                             comm_stream_));

  CUDA_CHECK(hipMemcpyAsync(rdma_buffer_ptr_, tensor.data_ptr(),
                             tensor.nbytes(), hipMemcpyDeviceToDevice,
                             comm_stream_));
  // intranode barrier
  sync::barrier(barrier_signal_ptrs_gpu_, rank_, num_ranks_, comm_stream_);

  // internode barrier
  nvshmem::barrier();

  for (int rank = 0; rank < num_ranks_; ++rank) {
    // TODO(KuangjuX): Implement this
    bool is_same_node = true;

    if (is_same_node) {
      // Intranode: CUDA IPC
      CUDA_CHECK(hipMemcpyAsync(tensor_list[rank].data_ptr(),
                                 buffer_ptrs_[rank], tensor.nbytes(),
                                 hipMemcpyDeviceToDevice, comm_stream_));
    } else {
      // Internode: RDMA
      nvshmem::get_mem(tensor_list[rank].data_ptr(), rdma_buffer_ptr_,
                       tensor.nbytes(), rank);
    }

    if (!async_op) {
      hipStreamSynchronize(comm_stream_);
    }
  }
}

}  // namespace nvshmem_tutorial
