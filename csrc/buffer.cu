#include "hip/hip_runtime.h"
#include "buffer.cuh"

#include <cstring>

// Helper to get NVSHMEM local pe info
static inline void query_local_pe(int& local_pe, int& num_local_pes) {
  local_pe = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
  num_local_pes = nvshmem_team_n_pes(NVSHMEMX_TEAM_NODE);
}

Buffer::Buffer(int rank, int num_ranks, int64_t num_nvl_bytes,
               int64_t num_rdma_bytes)
    : rank_{rank},
      num_ranks_{num_ranks},
      num_nvl_bytes_{num_nvl_bytes},
      num_rdma_bytes_{num_rdma_bytes} {
  CUDA_CHECK(hipGetDevice(&device_id_));
  query_local_pe(local_pe_, num_local_pes_);

  rdma_rank_ = rank / NUM_MAX_NVL_PEERS;
  nvl_rank_ = rank % NUM_MAX_NVL_PEERS;
  num_rdma_ranks_ = std::max(1, num_ranks / NUM_MAX_NVL_PEERS);
  num_nvl_ranks_ = std::min(num_ranks, NUM_MAX_NVL_PEERS);

  // Get ranks
  CUDA_CHECK(hipGetDevice(&device_id_));

  if (num_nvl_bytes_ > 0) {
    // Local IPC: alloc local memory and set local IPC handles.
    CUDA_CHECK(hipMalloc(&buffer_ptrs_[nvl_rank_], num_nvl_bytes_));
    local_allocated_ = true;
    CUDA_CHECK(
        hipIpcGetMemHandle(&ipc_handles_[nvl_rank_], buffer_ptrs_[nvl_rank_]));
  }
}

Buffer::~Buffer() {
  // Close peer handles
  for (int i = 0; i < NUM_MAX_NVL_PEERS; ++i) {
    if (i != nvl_rank_ && buffer_ptrs_[i] != nullptr) {
      hipIpcCloseMemHandle(buffer_ptrs_[i]);
      buffer_ptrs_[i] = nullptr;
    }
  }
  if (local_allocated_ && buffer_ptrs_[nvl_rank_] != nullptr) {
    CUDA_CHECK(hipFree(buffer_ptrs_[nvl_rank_]));
    buffer_ptrs_[nvl_rank_] = nullptr;
  }
}

torch::Tensor Buffer::alloc_symmetric(int64_t size_bytes) {
  void* ptr = nvshmem_malloc(size_bytes);
  if (ptr == nullptr) {
    throw std::runtime_error("nvshmem_malloc returned nullptr");
  }
  return torch::from_blob(ptr, {size_bytes},
                          torch::dtype(torch::kUInt8).device(torch::kCUDA));
}

void Buffer::free_symmetric(torch::Tensor t) { nvshmem_free(t.data_ptr()); }

// py::bytearray Buffer::get_local_nvshmem_unique_id() const {
//   // Mimic DeepEP: only RDMA root (rdma_rank==0) allowed to export unique id
//   if (rdma_rank_ != 0) {
//     throw std::runtime_error("Only RDMA rank 0 can get NVSHMEM unique ID");
//   }
//   auto uid = get_unique_id();
//   return py::bytearray(reinterpret_cast<const char*>(uid.data()),
//   uid.size());
// }

// void Buffer::sync(
//     const std::vector<int>& device_ids,
//     const std::vector<std::optional<py::bytearray>>& all_gathered_handles,
//     const std::optional<py::bytearray>& root_unique_id_opt) {
//   // Open CUDA IPC peers
//   if (num_nvl_bytes_ > 0) {
//     if (static_cast<int>(device_ids.size()) != num_ranks_) {
//       throw std::runtime_error("sync: device_ids size mismatch");
//     }
//     if (static_cast<int>(all_gathered_handles.size()) != num_ranks_) {
//       throw std::runtime_error("sync: handles size mismatch");
//     }
//     // Map only peers within our NVL group
//     int offset = rdma_rank_ * num_nvl_ranks_;
//     for (int i = 0; i < num_nvl_ranks_; ++i) {
//       auto peer_rank = offset + i;
//       auto handle_str = std::string(
//           all_gathered_handles[peer_rank].value().cast<std::string>());
//       if (peer_rank != rank_) {
//         if (handle_str.size() != sizeof(hipIpcMemHandle_t)) {
//           throw std::runtime_error("CUDA IPC handle size mismatch");
//         }
//         std::memcpy(ipc_handles_[i].reserved, handle_str.data(),
//                     sizeof(hipIpcMemHandle_t));
//         CUDA_CHECK(hipIpcOpenMemHandle(&buffer_ptrs_[i], ipc_handles_[i],
//                                         hipIpcMemLazyEnablePeerAccess));
//       }
//     }
//     CUDA_CHECK(hipDeviceSynchronize());
//   }

//   // Initialize NVSHMEM and allocate RDMA buffer
//   if (num_rdma_bytes_ > 0) {
//     if (!root_unique_id_opt.has_value()) {
//       throw std::runtime_error("sync: missing root NVSHMEM unique id");
//     }
//     auto uid_str = root_unique_id_opt->cast<std::string>();
//     std::vector<int8_t> uid_vec(uid_str.begin(), uid_str.end());

//     int nvshmem_rank = rdma_rank_;
//     int nvshmem_world = num_rdma_ranks_;
//     init_with_unique_id(uid_vec, nvshmem_rank, nvshmem_world);

//     nvshmem_barrier_all();
//     rdma_buffer_ptr_ = nvshmem_malloc(num_rdma_bytes_);
//     CUDA_CHECK(hipMemset(rdma_buffer_ptr_, 0, num_rdma_bytes_));
//     nvshmem_barrier_all();
//   }

//   available_ = true;
// }

py::bytearray Buffer::get_local_ipc_handle() const {
  if (!local_allocated_ || num_nvl_bytes_ == 0) {
    throw std::runtime_error("No local NVLink buffer allocated");
  }
  return py::bytearray(
      reinterpret_cast<const char*>(ipc_handles_[nvl_rank_].reserved),
      sizeof(hipIpcMemHandle_t));
}

void Buffer::open_ipc_handles(
    const std::vector<std::optional<py::bytearray>>& all_handles) {
  if (num_nvl_bytes_ == 0) return;
  if (static_cast<int>(all_handles.size()) != num_local_pes_) {
    throw std::runtime_error(
        "open_ipc_handles: size mismatch with num_local_pes");
  }
  for (int pe = 0; pe < num_local_pes_; ++pe) {
    if (pe == local_pe_) continue;
    if (!all_handles[pe].has_value()) {
      throw std::runtime_error("Missing IPC handle for peer");
    }
    auto h = std::string(all_handles[pe]->cast<std::string>());
    if (h.size() != sizeof(hipIpcMemHandle_t)) {
      throw std::runtime_error("IPC handle size mismatch");
    }
    std::memcpy(ipc_handles_[pe].reserved, h.data(),
                sizeof(hipIpcMemHandle_t));
    CUDA_CHECK(hipIpcOpenMemHandle(&buffer_ptrs_[pe], ipc_handles_[pe],
                                    hipIpcMemLazyEnablePeerAccess));
  }
}

// void Buffer::intranode_memcpy_to(int dst_local_pe, int64_t dst_offset_bytes,
//                                  torch::Tensor src) {
//   if (dst_local_pe < 0 || dst_local_pe >= num_local_pes_) {
//     throw std::runtime_error("dst_local_pe out of range");
//   }
//   if (buffer_ptrs_[dst_local_pe] == nullptr) {
//     throw std::runtime_error("Destination peer buffer not mapped");
//   }
//   if (!src.is_cuda()) {
//     throw std::runtime_error("src must be CUDA tensor");
//   }
//   int64_t nbytes = src.nbytes();
//   void* dst =
//       static_cast<uint8_t*>(buffer_ptrs_[dst_local_pe]) + dst_offset_bytes;
//   CUDA_CHECK(hipMemcpy(dst, src.data_ptr(), nbytes,
//   hipMemcpyDeviceToDevice));
// }

torch::Tensor Buffer::get_local_buffer_u8() const {
  if (!local_allocated_ || num_nvl_bytes_ == 0) {
    throw std::runtime_error("No local NVLink buffer allocated");
  }
  return torch::from_blob(buffer_ptrs_[nvl_rank_], {num_nvl_bytes_},
                          torch::dtype(torch::kUInt8).device(torch::kCUDA));
}

// intranode communication kernel

void Buffer::intranode_all_to_all(torch::Tensor input, torch::Tensor output,
                                  torch::Tensor input_split_sizes,
                                  torch::Tensor output_split_sizes) {}

void Buffer::internode_put(torch::Tensor dst_symmetric, torch::Tensor src,
                           int dst_pe) {
  if (!src.is_cuda() || !dst_symmetric.is_cuda()) {
    throw std::runtime_error("internode_put expects CUDA tensors");
  }
  nvshmem_putmem(dst_symmetric.data_ptr(), src.data_ptr(), src.nbytes(),
                 dst_pe);
  nvshmem_quiet();
}

void Buffer::internode_get(torch::Tensor dst, torch::Tensor src_symmetric,
                           int src_pe) {
  if (!dst.is_cuda() || !src_symmetric.is_cuda()) {
    throw std::runtime_error("internode_get expects CUDA tensors");
  }
  nvshmem_getmem(dst.data_ptr(), src_symmetric.data_ptr(), dst.nbytes(),
                 src_pe);
  nvshmem_quiet();
}

void Buffer::internode_all_to_all(torch::Tensor input, torch::Tensor output,
                                  torch::Tensor input_split_sizes,
                                  torch::Tensor output_split_sizes) {}

torch::Tensor Buffer::get_local_buffer_tensor(const py::object& dtype,
                                              int64_t offset,
                                              bool use_rdma_buffer) const {
  auto scalar_type = torch::python::detail::py_object_to_dtype(dtype);
  int64_t elem_size = c10::elementSize(scalar_type);
  void* base_ptr = use_rdma_buffer ? rdma_buffer_ptr_ : buffer_ptrs_[nvl_rank_];
  int64_t num_bytes = use_rdma_buffer ? num_rdma_bytes_ : num_nvl_bytes_;
  if (base_ptr == nullptr || num_bytes == 0) {
    throw std::runtime_error("Requested buffer is not available");
  }
  auto ptr = static_cast<uint8_t*>(base_ptr) + offset;
  return torch::from_blob(
      ptr, num_bytes / elem_size,
      torch::TensorOptions().dtype(scalar_type).device(torch::kCUDA));
}

void Buffer::destroy() {
  CUDA_CHECK(hipDeviceSynchronize());
  // Close CUDA IPC
  for (int i = 0; i < NUM_MAX_NVL_PEERS; ++i) {
    if (i != nvl_rank_ && buffer_ptrs_[i] != nullptr) {
      hipIpcCloseMemHandle(buffer_ptrs_[i]);
      buffer_ptrs_[i] = nullptr;
    }
  }
  if (local_allocated_ && buffer_ptrs_[nvl_rank_] != nullptr) {
    CUDA_CHECK(hipFree(buffer_ptrs_[nvl_rank_]));
    buffer_ptrs_[nvl_rank_] = nullptr;
  }
  // Free NVSHMEM
  if (rdma_buffer_ptr_ != nullptr) {
    nvshmem_barrier_all();
    nvshmem_free(rdma_buffer_ptr_);
    rdma_buffer_ptr_ = nullptr;
  }
  available_ = false;
}

int Buffer::get_local_pe() const { return local_pe_; }
int Buffer::get_num_local_pes() const { return num_local_pes_; }
int Buffer::get_local_device_id() const { return device_id_; }
int64_t Buffer::get_num_nvl_bytes() const { return num_nvl_bytes_; }